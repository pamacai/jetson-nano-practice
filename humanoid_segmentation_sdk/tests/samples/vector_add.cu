#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main() {
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = i * 0.5f;
        h_B[i] = i * 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "C[100] = " << h_C[100] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
